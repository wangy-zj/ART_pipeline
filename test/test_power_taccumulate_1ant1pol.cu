#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

/*
  This is the main function to test krnl_power_taccumulate_1ant1pol kernel.
  the kernel does not produce dig value?
*/

#include "../include/cuda/cuda_utilities.h"
#include "../include/test.h"
#include "../include/krnl.h"

#include <iostream> // For cout

#define TSAMP_PKT 8.192E-3 // milliseconds
#define NSAMP_PKT 8192

#define TIMEIT

#ifdef TIMEIT
#define CUDA_STARTTIME(x)  hipEventRecord(x ## _start, 0);

#define CUDA_STOPTIME(x) {					\
    float dtime;						\
    hipEventRecord(x ## _stop, 0);				\
    hipEventSynchronize(x ## _stop);				\
    hipEventElapsedTime(&dtime, x ## _start, x ## _stop);	\
    x ## time += dtime; }

#else
#define CUDA_STARTTIME(x)
// It is better to sync even we do not timing information
#define CUDA_STOPTIME(x)
#endif

#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "../doctest/doctest.h"

TEST_CASE("power_taccumulate_1ant1pol") {
  
  // Setup
  int gpu = 0;
  //int npkt = 8192;
  int npkt = 8192;
  int nthread = 128;
  int nfft_point = 8192;
  
  int nchan = nfft_point/2+1;
  int nfft  = npkt*NSAMP_PKT/nfft_point;
  int ninput = nfft*nchan;

  int reset = 1;
  
  fprintf(stdout, "DEBUG: gpu = %d\n", gpu);
  fprintf(stdout, "DEBUG: npkt = %d\n", npkt);
  fprintf(stdout, "DEBUG: nfft = %d\n", nfft);
  fprintf(stdout, "DEBUG: nchan = %d\n", nchan);
  fprintf(stdout, "DEBUG: nthread = %d\n", nthread);
  
  // Setup GPU with ID and print out its name
  hipDeviceProp_t prop = {0};
  int gpu_get = gpuDeviceInit(gpu); // The required gpu might be different from what we get
  fprintf(stdout, "Asked for GPU %d, got GPU %d\n", gpu, gpu_get);
  checkCudaErrors(hipGetDeviceProperties(&prop, gpu_get));
  fprintf(stdout, "GPU name is %s\n", prop.name);

  // Setup buffers  
  float mean = 0;
  float stddev = 6400;
  hiprandGenerator_t gen;
  checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

  // data
  RealGeneratorNormal data_r(gen, mean, stddev, ninput);
  RealGeneratorNormal data_i(gen, mean, stddev, ninput);
  ComplexBuilder<float, float, hipComplex> data(data_r.data, data_i.data, ninput, nthread);
  
  // Setup timer for both CPU and GPU
#ifdef TIMEIT
  // Setup CPU timer
  struct timespec c_start = {0};
  struct timespec c_stop  = {0};
  double c_elapsed = 0;
  
  // Setup GPU timer
  hipEvent_t g_start = {0};
  hipEvent_t g_stop  = {0};
  float gtime = 0;
  checkCudaErrors(hipEventCreate(&g_start));
  checkCudaErrors(hipEventCreate(&g_stop));
  clock_gettime(CLOCK_REALTIME, &c_start);
#endif
  
  // Do processing on CPU
  ManagedMemoryAllocator<float> h_result(nchan);
  power_taccumulate_1ant1pol((CPP_COMPLEX*)data.data, h_result.data, nfft, nchan, reset);
  
#ifdef TIMEIT
  clock_gettime(CLOCK_REALTIME, &c_stop);
  c_elapsed = (c_stop.tv_sec - c_start.tv_sec) +
    (c_stop.tv_nsec - c_start.tv_nsec)/1.0E9L;

  fprintf(stdout, "tsamp_pkt is %f milliseconds, we have %d packets, available time is %f milliseconds\n", TSAMP_PKT, npkt, TSAMP_PKT*npkt);
  fprintf(stdout, "elapsed_time for CPU version is %f milliseconds\n", 1.0E3*c_elapsed);
#endif
  
  fprintf(stdout, "Finish CPU execution\n");
  
  // Now run kernel on GPU
  ManagedMemoryAllocator<float> d_result(nchan);
  
#ifdef TIMEIT
  CUDA_STARTTIME(g);
#endif

  krnl_power_taccumulate_1ant1pol<<<nchan/nthread+1, nthread>>>(data.data, d_result.data, nfft, nchan, reset);
  getLastCudaError("Kernel execution failed [ krnl_power_taccumulate_1ant1pol ]");
  
#ifdef TIMEIT
  // CUDA_STOPTIME has sync inside
  CUDA_STOPTIME(g);
  fprintf(stdout, "elapsed time for krnl_power_taccumulate_1ant1pol running on %s is %f milliseconds\n\n", prop.name, gtime);
#else
  checkCudaErrors(hipDeviceSynchronize());
#endif

  // Now check numbers as complex
  RealDifferentiator<float, float> diff(d_result.data, h_result.data, nchan, nthread);
  
  RealMeanStddevCalculator<float> mean_stddev_diff(diff.data, nchan, nthread, 7);

  for(int i = 0; i < nchan; i++){
    //if(h_result.data[i]!=h_result.data[i])
      //std::cout << i << "\t" << h_result.data[i] << "\t" << h_result.data[i] << std::endl;
      fprintf(stdout, "%d\t%f\t%f\t%E\n", i, h_result.data[i], d_result.data[i], (h_result.data[i]-d_result.data[i])/h_result.data[i]);
  }
  std::cout << "\n";
  std::cout << mean_stddev_diff.mean << "\t" << mean_stddev_diff.stddev << std::endl;
  std::cout << std::endl;

  // Free allocated memory
  checkCudaErrors(hiprandDestroyGenerator(gen));
}
