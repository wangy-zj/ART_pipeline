#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

/*
  This is the main function to test krnl_tf2ft_1ant1pol kernel.
  the kernel does not produce dig value?
*/

#include "../include/cuda/cuda_utilities.h"
#include "../include/test.h"
#include "../include/krnl.h"

#include <iostream> // For cout

#define TSAMP_PKT 8.192E-3 // milliseconds
#define NSAMP_PKT 8192

#define TIMEIT

#ifdef TIMEIT
#define CUDA_STARTTIME(x)  hipEventRecord(x ## _start, 0);

#define CUDA_STOPTIME(x) {					\
    float dtime;						\
    hipEventRecord(x ## _stop, 0);				\
    hipEventSynchronize(x ## _stop);				\
    hipEventElapsedTime(&dtime, x ## _start, x ## _stop);	\
    x ## time += dtime; }

#else
#define CUDA_STARTTIME(x)
// It is better to sync even we do not timing information
#define CUDA_STOPTIME(x)
#endif

#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "../doctest/doctest.h"

TEST_CASE("tf2ft_1ant1pol") {
  
  // Setup
  int gpu = 0;
  int npkt = 8192;
  int nthread = 128;
  int nfft_point = 8192;
  
  int nchan = nfft_point/2+1;
  int nfft  = npkt*NSAMP_PKT/nfft_point;
  int nsamp = nfft*nchan;
  
  fprintf(stdout, "DEBUG: gpu = %d\n", gpu);
  fprintf(stdout, "DEBUG: npkt = %d\n", npkt);
  fprintf(stdout, "DEBUG: nfft = %d\n", nfft);
  fprintf(stdout, "DEBUG: nchan = %d\n", nchan);
  fprintf(stdout, "DEBUG: nthread = %d\n", nthread);
  
  // Setup GPU with ID and print out its name
  hipDeviceProp_t prop = {0};
  int gpu_get = gpuDeviceInit(gpu); // The required gpu might be different from what we get
  fprintf(stdout, "Asked for GPU %d, got GPU %d\n", gpu, gpu_get);
  checkCudaErrors(hipGetDeviceProperties(&prop, gpu_get));
  fprintf(stdout, "GPU name is %s\n", prop.name);

  // Setup buffers  
  float mean = 0;
  float stddev = 10;
  hiprandGenerator_t gen;
  checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

  // data
  RealGeneratorNormal data_r(gen, mean, stddev, nsamp);
  RealGeneratorNormal data_i(gen, mean, stddev, nsamp);
  ComplexBuilder<float, float, hipComplex> data(data_r.data, data_i.data, nsamp, nthread);
  
  // Setup timer for both CPU and GPU
#ifdef TIMEIT
  // Setup CPU timer
  struct timespec c_start = {0};
  struct timespec c_stop  = {0};
  double c_elapsed = 0;
  
  // Setup GPU timer
  hipEvent_t g_start = {0};
  hipEvent_t g_stop  = {0};
  float gtime = 0;
  checkCudaErrors(hipEventCreate(&g_start));
  checkCudaErrors(hipEventCreate(&g_stop));
  clock_gettime(CLOCK_REALTIME, &c_start);

#endif
  
  // Do processing on CPU
  ManagedMemoryAllocator<CPP_COMPLEX> h_result(nsamp);
  tf2ft_1ant1pol((CPP_COMPLEX*)data.data, h_result.data, nfft, nchan);
  
#ifdef TIMEIT
  clock_gettime(CLOCK_REALTIME, &c_stop);
  c_elapsed = (c_stop.tv_sec - c_start.tv_sec) +
    (c_stop.tv_nsec - c_start.tv_nsec)/1.0E9L;

  fprintf(stdout, "tsamp_pkt is %f milliseconds, we have %d packets, available time is %f milliseconds\n", TSAMP_PKT, npkt, TSAMP_PKT*npkt);
  fprintf(stdout, "elapsed_time for CPU version is %f milliseconds\n", 1.0E3*c_elapsed);
#endif
  
  fprintf(stdout, "Finish CPU execution\n");
  
  // Now run kernel on GPU
  dim3 grid_size(ceil(nchan/(double)TILE_DIM), ceil(nfft/(double)TILE_DIM), 1);
  dim3 block_size(TILE_DIM, NROWBLOCK_TRANS, 1);

  ManagedMemoryAllocator<hipComplex> d_result(nsamp);
  print_cuda_memory_info();
  
#ifdef TIMEIT
  CUDA_STARTTIME(g);
#endif

  krnl_tf2ft_1ant1pol<<<grid_size, block_size>>>(data.data, d_result.data, nfft, nchan);
  getLastCudaError("Kernel execution failed [ krnl_tf2ft_1ant1pol ]");
  
#ifdef TIMEIT
  // CUDA_STOPTIME has sync inside
  CUDA_STOPTIME(g);
  fprintf(stdout, "elapsed time for krnl_tf2ft_1ant1pol running on %s is %f milliseconds\n\n", prop.name, gtime);
#else
  checkCudaErrors(hipDeviceSynchronize());
#endif

  // Now check numbers as complex
  RealDifferentiator<float, float> diff((float*)d_result.data, (float*)d_result.data, 2*nsamp, nthread);
  
  RealMeanStddevCalculator<float> mean_stddev_diff(diff.data, 2*nsamp, nthread, 7);
  
  std::cout << "\n";
  std::cout << mean_stddev_diff.mean << "\t" << mean_stddev_diff.stddev << std::endl;
  std::cout << std::endl;

  // Free allocated memory
  checkCudaErrors(hiprandDestroyGenerator(gen));
}
